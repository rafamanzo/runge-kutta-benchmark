#include <cstdlib>
#include <cstdio>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <rkb_types.h>
#include <DataSet.h>
#include <Fiber.h>
#include <Fixture.h>
#include <CUDAStraightFixture.h>

using namespace RungeKuttaBenchmark;

CUDAStraightFixture::CUDAStraightFixture(vector *v0, int v0_count, DataSet dataset){
  size_t available, total;

  _v0_count = v0_count;
  _data_set = dataset;

  if(hipMalloc(&_v0, _v0_count*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the initial points\n", (_v0_count*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  if(hipMalloc(&_field, dataset.n_x()*dataset.n_y()*dataset.n_z()*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the vector field\n", (dataset.n_x()*dataset.n_y()*dataset.n_z()*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  if(hipMalloc(&_points_count, _v0_count*sizeof(int)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the points count vector\n", (_v0_count*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  hipMemGetInfo(&available, &total);
  _max_points = ((available*0.9)/(sizeof(vector)*_v0_count));
  if(hipMalloc(&_points, _v0_count*_max_points*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the fibers\n", (_v0_count*_max_points*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }

  if(hipMemcpy(_v0, v0, _v0_count*sizeof(vector), hipMemcpyHostToDevice) != hipSuccess){
    printf("Failed to transfer the initial points list to device\n");
    exit(-1);
  }

  if(hipMemcpy(_field, dataset.field(), dataset.n_x()*dataset.n_y()*dataset.n_z()*sizeof(vector), hipMemcpyHostToDevice) != hipSuccess){
    printf("Failed to transfer the vector field to the device\n");
    exit(-1);
  }
}

CUDAStraightFixture::~CUDAStraightFixture(){
  hipFree(_field);
  hipFree(_v0);
  hipFree(_points);
  hipFree(_points_count);
}

vector *CUDAStraightFixture::getPoints(){
  return _points;
}

int *CUDAStraightFixture::getPointsCount(){
  return _points_count;
}

int CUDAStraightFixture::getMaxPoints(){
  return _max_points;
}

vector_field CUDAStraightFixture::getField(){
  return _field;
}