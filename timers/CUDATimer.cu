#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <Timer.h>
#include <CUDATimer.h>

using namespace RungeKuttaBenchmark;

CUDATimer::CUDATimer(){
  hipEventCreate(&_proc_start);
  hipEventCreate(&_proc_finish);
  hipEventCreate(&_memo_start);
  hipEventCreate(&_memo_finish);
}

CUDATimer::~CUDATimer(){
  hipEventDestroy(_proc_start);
  hipEventDestroy(_proc_finish);
  hipEventDestroy(_memo_start);
  hipEventDestroy(_memo_finish);
}

void CUDATimer::startRecordProcTime(){
  hipEventRecord(_proc_start, 0);
}

void CUDATimer::stopRecordProcTime(){
  float time;

  hipEventRecord(_proc_finish, 0);
  hipEventSynchronize(_proc_finish);

  hipEventElapsedTime(&time, _proc_start, _proc_finish);

  _proc_time_in_sec += (double) (time/1000.0);
}

void CUDATimer::startRecordMemoTime(){
  hipEventRecord(_memo_start, 0);
}

void CUDATimer::stopRecordMemoTime(){
  float time;

  hipEventRecord(_memo_finish, 0);
  hipEventSynchronize(_memo_finish);

  hipEventElapsedTime(&time, _memo_start, _memo_finish);

  _memo_time_in_sec += (double) (time/1000.0);
}